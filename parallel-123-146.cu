/*
	* Team - Suraj Singh and Mahir Jain 
	* Roll Numbers - 16CO146 and 16CO123 respectively.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define r_size 10
#define c_size 28
// Only one black is used.
#define BLOCKS 1
// Depends on the GPU used.
#define THREADS 1024


// Function for calculating execution time
void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %fs\n", elapsed);
}


// Main CUDA kernel
// most of the parameters here are too big to be stored in shared memory
// and hence we have used global memory instead
__global__ void brandes(int s, int *R, int *C, int  *S, int *d, float *sigma, float *delta, int *Q, int *Q2, int * ends, float *bc) {
    
    int idx = threadIdx.x;
    // Initialise values for BFS
    for(int k=idx; k < r_size; k+= blockDim.x) {
        //printf("hi");
        if( k == s ) {
            d[k] = 0;
            sigma[k] =1;
        } else {
            d[k] = -1;
            sigma[k] = 0;
        }
    }
	// initialize variables common to all threads in the block that fit in shared memory
    __shared__ int Q_len;
    __shared__ int Q2_len;
    __shared__ int ends_len;
    __shared__ int depth;
    __shared__ int S_len;
    int count;
 
    if( idx == 0 ) {
        Q[0] = s;
        Q_len = 1;
        S[0] = s;
        S_len=1;
        Q2_len = 0;
        ends[0] =0;
        ends[1] = 1;
        ends_len = 2;
    }

    __syncthreads();

    count =0;
    while(1) {
        for(int k=idx; k < ends[count+1] - ends[count]; k+=blockDim.x) {
            int v = Q[k];            
            // Same logic as ocean kernel! 
            __syncthreads();
            for(int r = R[v]; r< R[v+1]; r++) {
                int w = C[r];
            
                int t;
                // Adding neighbours to our 'stack' implemented as a queue
                if (atomicCAS(&d[w], -1, d[v]+1) == -1) {
                    //printf("%d\n", w);
                    t = atomicAdd(&Q2_len,1);
                        //f =1;
                    Q2[t] = w;
                }
				// if v was the shortest path to w, update sigma
                
                if(d[w] == (d[v]+1)) {
                    atomicAdd(&sigma[w],sigma[v]);
                }
            }
        }
        __syncthreads();

        if(Q2_len==0) {
            if(idx==0) {
            // calculate depth for next section of code
                depth = d[S[S_len-1]];
            }
            break;
        } else {
        // swap Q with Q2
            for(int k =idx; k < Q2_len; k+=blockDim.x) {
                Q[k] = Q2[k];
                S[k+S_len]  = Q2[k];
            }
            __syncthreads();
            if(idx==0) {
                ends[ends_len] = ends[ends_len-1] + Q2_len;
                ends_len = ends_len +1;
                Q_len = Q2_len;
                S_len = S_len + Q2_len;
                Q2_len = 0;
            }
            __syncthreads();
            

        }

        count++;
        __syncthreads();

    }
    // everyone needs to stop after breaking out of while loop
    __syncthreads();
    while(depth > 0) {
    // all threads execute in parallel
        if(idx >= ends[depth] && idx <= ends[depth+1] -1)
        {
            int w = S[idx];
            float dsw = 0;
            float sw = sigma[w];
            // update delta for a vertex by traversing its neighbours 
            for(int r = R[w]; r< R[w+1]; r++) {
                int v = C[r];
                if(d[v] == d[w] + 1) {
                    dsw += (sw/sigma[v])*(1 + delta[v]);
                }
            }
            delta[w] = dsw;
            __syncthreads();
            // add to BC value of the vertex!
            if(w!=s) {
                atomicAdd(&bc[w],delta[w]/2);
                //bc[w] += delta[w]/2;
            }
        }
        depth--;
    }


}


int main(int argc, char const *argv[])
{
    FILE *R = fopen("R.txt", "r");
    FILE *C = fopen("C.txt", "r");
    clock_t start, stop;
    int r[r_size];
    int c[c_size];
    for(int i=0;i< r_size; i++) {
        fscanf(R, "%d\n", &r[i]);
    }
    for(int i=0;i< c_size; i++) {
        fscanf(C, "%d\n", &c[i]);
    }

    int *dev_r, *dev_c, *dev_d, *dev_Q, *dev_Q2,*ends, *dev_S;

    float *dev_sigma, *dev_delta, *dev_bc;
    float *bc_val = (float*)malloc(r_size*sizeof(float));


    hipMalloc((void**) &dev_r, r_size*sizeof(int));
    hipMalloc((void**) &dev_c, c_size*sizeof(int));
    hipMalloc((void**) &dev_bc, r_size*sizeof(float));
    hipMalloc((void**) &dev_d, r_size*sizeof(int));
    hipMalloc((void**) &dev_sigma, r_size*sizeof(float));
    hipMalloc((void**) &dev_delta, r_size*sizeof(float));
    hipMalloc((void**) &dev_Q, r_size*sizeof(int));
    hipMalloc((void**) &dev_Q2, r_size*sizeof(int));
    hipMalloc((void**) &dev_S, r_size*sizeof(int));
    hipMalloc((void**) &ends, (r_size+1)*sizeof(int));


    hipMemcpy(dev_r, r, r_size*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, c_size*sizeof(int), hipMemcpyHostToDevice);


    dim3 blocks(BLOCKS,1);
    dim3 threads(THREADS,1);

    start = clock();
    for(int s=0; s < r_size; s++) {
        brandes<<<blocks, threads>>>(s, dev_r, dev_c, dev_S, dev_d , dev_sigma, dev_delta,dev_Q,dev_Q2, ends, dev_bc);
     }
    stop=clock();
    
    print_elapsed(start,stop);
    hipMemcpy(bc_val, dev_bc, r_size*(sizeof(float)), hipMemcpyDeviceToHost);

   


    return 0;
}
